#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6


__global__ void vector_add(float *out, float *a, float *b, int n){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
        //aprintf("%f",out[i]);
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Memory Alocation
    a = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    // cudaMalloc(void **devPtr, size_t count);
    // devPtr -> Device pointer 
    // count -> memory  size
    // cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind)
    //

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);


    // Transfer data from host to device Memory

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,256>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a); 
    free(b); 
    free(out);

    return 0;
}