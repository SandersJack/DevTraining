
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n){
    for(int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
        //aprintf("%f",out[i]);
    }
}

int main() {
    float *a, *b, *out;
    float *d_a;

    // Memory Alocation
    a = (float*)malloc(sizeof(float) * N);

    // Allocate device memory
    // cudaMalloc(void **devPtr, size_t count);
    // devPtr -> Device pointer 
    // count -> memory  size
    // cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind)
    //

    hipMalloc((void**)&d_a, sizeof(float)*N);

    // Transfer data from host to device Memory

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1,1>>>(out, d_a, b, N);

    hipFree(d_a);
    free(a);
    return 0;
}